
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define FILENAME "./dblp-co-authors.txt"
#define NumAuthor 317080
#define DataLen   1049866
#define BlockSize 1024
#define GridSize  int(DataLen/BlockSize) + 1

int dataset[DataLen * 2];// array to store the raw dataset

void dataset_read(int * dataset);

__global__ void dataset_parse(int * dataset, int * output);

int dataset_maxCoAuthor(int * output, int lenght);

void dataset_plot(int * output, int lenght, int max);

int main(int argc, char * argv[])
{
    int output[NumAuthor] = { 0 };
    int * cu_output;//array to store the co-authors number of each author

    dataset_read(dataset);

    // Set device that we will use for our cuda code
    hipSetDevice(0);
    
    // Time Variables
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    float time;
    
    int * cu_dataset;
    hipEventRecord(start,0);
    hipMalloc((void**)&cu_output, NumAuthor * sizeof(int) );
    hipMalloc((void**)&cu_dataset, DataLen * 2 * sizeof(int));
    hipMemcpy(cu_dataset, dataset, DataLen * 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cu_output, output, NumAuthor  * sizeof(int), hipMemcpyHostToDevice);
    
    dataset_parse<<<GridSize, BlockSize>>>(cu_dataset, cu_output);
    hipDeviceSynchronize();
    //cudaEventSynchronize(stop);

    //cudaEventElapsedTime(&time, start, stop);
    hipMemcpy(output, cu_output, NumAuthor  * sizeof(int), hipMemcpyDeviceToHost);
   
    hipEventRecord(stop,0);
    hipEventElapsedTime(&time, start, stop);  
    int max = dataset_maxCoAuthor(output, NumAuthor);
    printf("Time elapsed: %f\n", time);
    dataset_plot(output, NumAuthor, max);
    return 0;
}


void dataset_read( int * dataset)
{
    FILE * datafile;
    datafile = fopen( FILENAME, "r");
    char line[255];
    while (true)
    {
        fscanf(datafile, "%s", line);
        if (atoi(line) == 1)
        {
            dataset[0] = 1;
            break;
        }
    }
    for(int i = 1; i < NumAuthor; i++){
        fscanf(datafile, "%d", &dataset[i]);
    }
    fclose(datafile);
}

__global__ void dataset_parse(int * dataset, int * output)
{
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
     if(indx < DataLen){   
        atomicAdd(&(output[dataset[2*indx]-1]), 1);
        atomicAdd(&(output[dataset[2*indx+1]-1]), 1);
        //if (dataset[2*indx]-1 >= 315280)
        //    printf("index: %6d author:%6d output:%6d\n", indx,dataset[2*indx]-1, output[dataset[2*indx]-1]);
        //if (dataset[2*indx+1]-1 >= 315280)
        //    printf("index: %6d author:%6d output:%6d\n", indx,dataset[2*indx+ 1]-1, output[dataset[2*indx+1]-1]);
}
}

int dataset_maxCoAuthor(int * output, int lenght)
{
    int max =0;
    int max_num = 0;
    int max_ind[1000] = { 0 };
    //memset(max_ind, 0, 1000);
    for(int i = 0; i < lenght; i++)
    {
        //printf("output:%d, %d", i, output[i]);
        if(max < output[i])
        {
           // printf("Max right now:%d, %d\n", i, output[i]);
            max = output[i];
            max_num = 0;
            memset(max_ind, 0, 1000);
            max_ind[max_num] = i;
        }
        else if(max == output[i])
        {
            max_num++;
            max_ind[max_num] = i;
        }
        //else{
        //printf("max is:%d, %d\n", max, max_ind[0]);
         //}
    }
    printf("The  list of authors with most co-authors:\n");
    for(int i = 0; i <= max_num; i++)
    {
    printf("Author: %6d has %6d co-authors.\n", max_ind[i] + 1, output[max_ind[i]]);
    }
    return output[max_ind[0]];
}

void dataset_plot(int * output, int lenght, int max)
{
    //int* numCoAuthorList;
    int* numCoAuthorList = (int*)malloc(max * sizeof(int));
    memset(numCoAuthorList, 0, sizeof(numCoAuthorList));
    for(int i = 0; i < lenght; i++)
    {
        if(output[i] <= max)
        {
            numCoAuthorList[output[i] - 1]++;
        }
        else{
            printf("\nError in Finding MAX!!!\n");
        }
    }
    FILE *fp;

    fp = fopen("./output.txt", "wb");

    fwrite(numCoAuthorList, sizeof(int), sizeof(numCoAuthorList), fp);
    fclose(fp);

}
