
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define FILENAME "./dblp-co-authors.txt"
#define NumAuthor 317080
#define DataLen   1049866
#define BlockSize 1024 
#define GridSize  int(DataLen/BlockSize) + 1
#define MAX 343
#define newGridSize int(NumAuthor/BlockSize) + 1
int dataset[DataLen * 2];// array to store the raw dataset

void dataset_read(int * dataset);
__global__ void dataset_parse(int * dataset, int * output,  int * full_output);
//int dataset_maxCoAuthor(int * output, int lenght);
//void dataset_plot(int * output, int lenght, int max);
__global__ void output_parse(int * full_output, int * output, int * num_author_array);
__device__ void check_pair(int * full_output, int start, int len, int possible_pair, int * pair_array, int * pair_len);
__device__ void count_diff_auth(int * pair_array, int * new_array,int pair_len, int * pure_len, int indx);

int main(int argc, char * argv[])
{
    int output[NumAuthor] = {0};
    int full_output[MAX * NumAuthor] = { 0 };
    int * cu_output;//array to store the co-authors number of each author
    int * cu_full_output;

    dataset_read(dataset);

    // Set device that we will use for our cuda code
    hipSetDevice(0);
    
    // Time Variables
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    float time;
    
    int * cu_dataset;
    hipEventRecord(start,0);
    hipMalloc((void**)&cu_output, NumAuthor * sizeof(int));
    hipMalloc((void**)&cu_full_output, MAX * NumAuthor * sizeof(int));
    hipMalloc((void**)&cu_dataset, DataLen * 2 * sizeof(int));
    hipMemcpy(cu_dataset, dataset, DataLen * 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cu_full_output, full_output, MAX * NumAuthor  * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cu_output, output, NumAuthor  * sizeof(int), hipMemcpyHostToDevice);

    dataset_parse<<<GridSize, BlockSize>>>(cu_dataset, cu_output, cu_full_output);
    hipDeviceSynchronize();
    hipMemcpy(output, cu_output, NumAuthor  * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(full_output, cu_full_output, MAX * NumAuthor  * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);  
    //int max = dataset_maxCoAuthor(output, NumAuthor);
    //printf("Time elapsed: %f\n", time);
    
    int * cu_num_author_array;
    int * num_author_array = (int*)malloc(NumAuthor * sizeof(int));
    hipMalloc((void**)&cu_num_author_array, NumAuthor * sizeof(int));
    hipMemset(cu_num_author_array, 0, NumAuthor * sizeof(int));
    
    hipEventRecord(start,0);
    output_parse<<<newGridSize, BlockSize>>>(cu_full_output, cu_output, cu_num_author_array); 
    hipDeviceSynchronize();
    printf("Error in Kernel output_parse:%s\n",hipGetErrorString(hipGetLastError()));    
    hipDeviceSynchronize();
    hipMemcpy(num_author_array, cu_num_author_array, NumAuthor  * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time in Kernel output_parse: %f\n", time);
    int total_number = 0;
    for (int i = 0; i < NumAuthor; i++)
         total_number += num_author_array[i];
    printf("Total number of authors is %d\n", total_number/2);
    return 0;
}

void dataset_read( int * dataset)
{
    FILE * datafile;
    datafile = fopen( FILENAME, "r");
    char line[255];
    while (true)
    {
        fscanf(datafile, "%s", line);
        if (atoi(line) == 1)
        {
            dataset[0] = 1;
            break;
        }
    }
    for(int i = 1; i < 2 * DataLen; i++){
        fscanf(datafile, "%d", &dataset[i]);
    }
    fclose(datafile);
}

__global__ void dataset_parse(int * dataset, int * output, int * full_output)
{
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    int i, j;
     if(indx < DataLen){   
        i = atomicAdd(&(output[dataset[2*indx]-1]), 1);
        full_output[(dataset[2*indx]-1) * MAX + i] = dataset[2*indx+1];         
        j = atomicAdd(&(output[dataset[2*indx+1]-1]), 1);
        full_output[(dataset[2*indx + 1]-1) * MAX + j] = dataset[2*indx];
}
}

/*
int dataset_maxCoAuthor(int * output, int lenght)
{
    int max =0;
    int max_num = 0;
    int max_ind[1000] = { 0 };
    //memset(max_ind, 0, 1000);
    for(int i = 0; i < lenght; i++)
    {
        //printf("output:%d, %d", i, output[i]);
        if(max < output[i])
        {
           // printf("Max right now:%d, %d\n", i, output[i]);
            max = output[i];
            max_num = 0;
            memset(max_ind, 0, 1000);
            max_ind[max_num] = i;
        }
        else if(max == output[i])
        {
            max_num++;
            max_ind[max_num] = i;
        }
        //else{
        //printf("max is:%d, %d\n", max, max_ind[0]);
         //}
    }
    printf("The  list of authors with most co-authors:\n");
    for(int i = 0; i <= max_num; i++)
    {
    printf("Author: %6d has %6d co-authors.\n", max_ind[i] + 1, output[max_ind[i]]);
    }
    return output[max_ind[0]];
}

void dataset_plot(int * output, int lenght, int max)
{
    //int* numCoAuthorList;
    int* numCoAuthorList = (int*)malloc(max * sizeof(int));
    memset(numCoAuthorList, 0, max);
    for(int i = 0; i < lenght; i++)
    {
        if(output[i] <= max)
        {
            numCoAuthorList[output[i] - 1]++;
        }
        else{
            printf("\nError in Finding MAX!!!\n");
        }
    }
    
    FILE *fp;

    fp = fopen("./output.txt", "wb");

    fwrite(numCoAuthorList, sizeof(int), max, fp);
    fclose(fp);

}
*/

__global__ void output_parse(int * full_output, int * output, int * num_author_array)
{
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    if(indx < NumAuthor){
        int pair_array[10000] = { 0 };
        int pair_len = 0;
        int coauthor, coauthor_co_len, possible_pair;
        for(int i = 0; i < output[indx]; i++){
            coauthor = full_output[indx * MAX + i];    
            coauthor_co_len = output[coauthor-1];
            for(int j = 0; j < coauthor_co_len; j++){
                possible_pair = full_output[(coauthor - 1) * MAX + j];
                check_pair(full_output, indx * MAX, output[indx], possible_pair, pair_array, &pair_len);
                }
        }

        //int * new_array = (int*)malloc(pair_len * sizeof(int));
        //memset(new_array, 0, pair_len );
        int new_array[10000] = {0};
        int num_authors = 0;
        count_diff_auth(pair_array, new_array, pair_len, &num_authors, indx);
        num_author_array[indx] = num_authors; 
    }
}

__device__ void check_pair(int * full_output, int start, int len, int possible_pair, int * pair_array, int * pair_len){
    for(int i = 0; i < len; i++){
        if(full_output[start + i] == possible_pair){
            pair_array[*pair_len] = possible_pair;
            (*pair_len)++;
            break;
        }        
    }         
}

__device__ void count_diff_auth(int * pair_array,int * new_array, int pair_len, int * pure_len, int indx){
    //int * new_array;
    //cudaMalloc((void**)&new_array, pair_len * sizeof(int));
    //cudaMemset(new_array, 0, pair_len * sizeof(int));
    //printf("pair_array[0]:%d, pair_len:%d, %d\n",pair_array[0], pair_len, indx);
    new_array[0] = pair_array[0];
    *pure_len = 1;
    for(int i = 1; i < pair_len; i++){
        int j;
        for(j = 0; j< *pure_len; j++){
            if(pair_array[i] == new_array[j]){
                break;
            }
        }
        
        if (j == *pure_len){
            new_array[*pure_len] = pair_array[i];
            (*pure_len)++;
        }  
    }
}
