
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add( int *a, int *b, int *c, int vector_size ) {
    
    // Calculate the index in the vector for the thread using the internal variables
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // HERE
    printf("%d\n", tid);    
    // This if statement is added in case we have more threads executing
    // Than number of elements in the vectors. How can this help?
    if (tid < vector_size){

    c[tid] = a[tid] + b[tid];    // HERE
        
    }
}

int main( int argc, char* argv[] ) { 

    // Parse Input arguments

    // Check the number of arguments
    if (argc != 3) {
        // Tell the user how to run the program
        printf ("Usage: %s vector_size block_size\n", argv[0]);
        // "Usage messages" are a conventional way of telling the user
        // how to run a program if they enter the command incorrectly.
        return 1;
    }
    
    // Set GPU Variables based on input arguments
    int vector_size = atoi(argv[1]);
    int block_size  = atoi(argv[2]);
    int grid_size   = ((vector_size-1)/block_size) + 1;

    // Set device that we will use for our cuda code
    hipSetDevice(0);
        
    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    // Input Arrays and variables
    int *a        = new int [vector_size]; 
    int *b        = new int [vector_size]; 
    int *c_cpu    = new int [vector_size]; 
    int *c_gpu    = new int [vector_size];

    // Pointers in GPU memory
    int *dev_a;
    int *dev_b;
    int *dev_c;

    // fill the arrays 'a' and 'b' on the CPU
    printf("Initializing input arrays.\n");
    for (int i = 0; i < vector_size; i++) {
        a[i] = rand()%10;
        b[i] = rand()%10;
    }

    //
    // CPU Calculation
    //////////////////

    printf("Running sequential job.\n");
    hipEventRecord(start,0);

    // Calculate C in the CPU
    for (int i = 0; i < vector_size; i++) {
            c_cpu[i] = a[i] + b[i];
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);

    // allocate the memory on the GPU
    hipMalloc(&dev_a, vector_size*sizeof(int)); // HERE
    hipMalloc(&dev_b, vector_size*sizeof(int));
    hipMalloc(&dev_c, vector_size*sizeof(int));

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, vector_size*sizeof(int), hipMemcpyHostToDevice);// HERE
    hipMemcpy(dev_b, b, vector_size*sizeof(int), hipMemcpyHostToDevice);
    
    //
    // GPU Calculation
    ////////////////////////

    printf("Running parallel job.\n");

    hipEventRecord(start,0);

    // call the kernel
    add<<<grid_size, block_size>>>(dev_a, dev_b, dev_c, vector_size); // HERE
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("\tParallel Job Time: %.2f ms\n", time);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c_gpu, dev_c, vector_size*sizeof(int), hipMemcpyDeviceToHost); // HERE (there's one more at the end, don't miss it!)
    
    // compare the results
    int error = 0;
    for (int i = 0; i < vector_size; i++) {
        if (c_cpu[i] != c_gpu[i]){
            error = 1;
            printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
        }
        if (error) break; 
    }

    if (error == 0){
        printf ("Correct result. No errors were found.\n");
    }

    // free CPU data
    free (a);
    free (b);
    free (c_cpu);
    free (c_gpu);

    // free the memory allocated on the GPU
    // HERE
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);    

    return 0;
}

